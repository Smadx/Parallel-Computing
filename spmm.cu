#include <iostream>
#include <hip/hip_runtime.h>

#include <cstddef>
#include <cstdlib>

using namespace std;

__global__ void spmm(int *dense, int *sparse, int *result, size_t pitch, int M, int N, int P, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < P)
    {
        int sum = 0;
        int *sparse_col = (int *)((char *)sparse + col * pitch);
        for (int i = 0; i < sparse_col[0]; i++)
        {
            sum += dense[row * N + sparse_col[i * 2 + 1]] * sparse_col[i * 2 + 2];
        }
        result[row * P + col] = sum;
    }
}

int main()
{
    int M, N, P, K;
    cin >> M >> N >> P >> K;
    int *dense = (int *)malloc(M * N * sizeof(int));
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            cin >> dense[i * N + j];
        }
    }

    // CSC
    int *sparse = (int *)malloc(P * (2 * K + 1) * sizeof(int));
    for (int i = 0; i < P; i++)
    {
        sparse[i * (2 * K + 1)] = 0;
    }
    for (int i = 0; i < K; i++)
    {
        int row, col, val;
        cin >> row >> col >> val;
        sparse[col * (2 * K + 1)]++;
        sparse[col * (2 * K + 1) + sparse[col * (2 * K + 1)] * 2 - 1] = row;
        sparse[col * (2 * K + 1) + sparse[col * (2 * K + 1)] * 2] = val;
    }

    int *result = (int *)malloc(M * P * sizeof(int));

    int *d_dense, *d_sparse, *d_result;
    hipMalloc(&d_dense, M * N * sizeof(int));
    size_t pitch;
    hipMallocPitch((void **)&d_sparse, &pitch, (2 * K + 1) * sizeof(int), P);
    hipMalloc(&d_result, M * P * sizeof(int));

    hipMemcpy(d_dense, dense, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy2D(d_sparse, pitch, sparse, (2 * K + 1) * sizeof(int), (2 * K + 1) * sizeof(int), P, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x, (P + threadsPerBlock.y - 1) / threadsPerBlock.y);

    spmm<<<numBlocks, threadsPerBlock>>>(d_dense, d_sparse, d_result, pitch, M, N, P, K);

    hipMemcpy(result, d_result, M * P * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_dense);
    hipFree(d_sparse);
    hipFree(d_result);

    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < P; j++)
        {
            cout << result[i * P + j] << " ";
        }
        cout << endl;
    }
    return 0;
}